
#include <iostream>
#include <hip/hip_runtime.h>


#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

#ifndef EPSILON_LOG_CUDA
#define EPSILON_LOG_CUDA 0.0000001f
#endif


__global__ void InitialGuessKernel(float* const __restrict__ solution, const float* const  __restrict__ rtm,
                                   const float* const  __restrict__ measured, const float* const  __restrict__  ray_density,
                                   const float ray_dens_thres, const size_t npixel, const size_t nvoxel){

    const size_t jvox = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t ipix_offset = blockIdx.y * BLOCK_SIZE;

    __shared__ float measured_cache[BLOCK_SIZE];

    const size_t cache_size = min(npixel - ipix_offset, (size_t)BLOCK_SIZE);
    if (threadIdx.x < cache_size) measured_cache[threadIdx.x] = measured[ipix_offset + threadIdx.x];

    __syncthreads();

    const float ray_dens = (jvox < nvoxel) ? ray_density[jvox] : 0;
    if (ray_dens > ray_dens_thres) {
        float res = 0;
        if (cache_size % 4 == 0) {
            for (size_t ipix_cache = 0; ipix_cache < cache_size; ipix_cache += 4) {
                const size_t ipix = ipix_offset + ipix_cache;
                res += rtm[ ipix      * nvoxel + jvox] * measured_cache[ipix_cache    ] + \
                       rtm[(ipix + 1) * nvoxel + jvox] * measured_cache[ipix_cache + 1] + \
                       rtm[(ipix + 2) * nvoxel + jvox] * measured_cache[ipix_cache + 2] + \
                       rtm[(ipix + 3) * nvoxel + jvox] * measured_cache[ipix_cache + 3];
            }
        }
        else {
            for (size_t ipix_cache = 0; ipix_cache < cache_size; ++ipix_cache) {
                res += rtm[(ipix_offset + ipix_cache) * nvoxel + jvox] * measured_cache[ipix_cache];
            }
        }
        res /= ray_dens;
        
        atomicAdd(solution + jvox, res);
    }
}


__global__ void PropagateKernel(float* const __restrict__ diff, const float* const __restrict__ rtm,
                                const float* const __restrict__ measured, const float* const __restrict__ fitted,
                                const float* const __restrict__ ray_density, const float* const __restrict__ ray_length,
                                const float* const __restrict__ grad_penalty, float relaxation, float ray_dens_thres, float ray_length_thres,
                                size_t npixel, size_t nvoxel){

    const size_t jvox = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t ipix_offset = blockIdx.y * blockDim.x;

    __shared__ float fit_diff_cache[BLOCK_SIZE];
    __shared__ float inv_length_cache[BLOCK_SIZE];

    const size_t cache_size = min(npixel - ipix_offset, (size_t)BLOCK_SIZE);
    if (threadIdx.x < cache_size) {
        const size_t ipix = ipix_offset + threadIdx.x;
        fit_diff_cache[threadIdx.x] = measured[ipix] - fitted[ipix];        
        const float length = ray_length[ipix];
        inv_length_cache[threadIdx.x] = (length > ray_length_thres) ? 1.f / length : 0;
    }

    __syncthreads();

    const float ray_dens = (jvox < nvoxel) ? ray_density[jvox] : 0;
    float res = 0;
    if (ray_dens > ray_dens_thres) {
        if (cache_size % 4 == 0) {
            for (unsigned int ipix_cache = 0; ipix_cache < cache_size; ipix_cache += 4) {
                const unsigned int ipix = ipix_offset + ipix_cache;
                res += rtm[ ipix      * nvoxel + jvox] * inv_length_cache[ipix_cache    ] * fit_diff_cache[ipix_cache    ] + \
                       rtm[(ipix + 1) * nvoxel + jvox] * inv_length_cache[ipix_cache + 1] * fit_diff_cache[ipix_cache + 1] + \
                       rtm[(ipix + 2) * nvoxel + jvox] * inv_length_cache[ipix_cache + 2] * fit_diff_cache[ipix_cache + 2] + \
                       rtm[(ipix + 3) * nvoxel + jvox] * inv_length_cache[ipix_cache + 3] * fit_diff_cache[ipix_cache + 3];
            }
        }
        else {
            for (unsigned int ipix_cache = 0; ipix_cache < cache_size; ++ipix_cache) {
                res += rtm[(ipix_offset + ipix_cache) * nvoxel + jvox] * inv_length_cache[ipix_cache] * fit_diff_cache[ipix_cache];
            }
        }
        res *= relaxation / ray_dens;
    }

    if (jvox < nvoxel) {
        if (!blockIdx.y) atomicAdd(diff + jvox, res - grad_penalty[jvox]);
        else atomicAdd(diff + jvox, res);
    }
}


__global__ void LogPropagateKernel(float* const __restrict__ obs_fit, const float* const __restrict__ rtm,
                                   const float* const __restrict__ measured, const float* const __restrict__ fitted,
                                   const float* const __restrict__ ray_density, const float* const __restrict__ ray_length,
                                   float ray_dens_thres, float ray_length_thres, size_t npixel, size_t nvoxel){

    const size_t jvox = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t ipix_offset = blockIdx.y * blockDim.x;

    __shared__ float measured_cache[BLOCK_SIZE];
    __shared__ float fitted_cache[BLOCK_SIZE];
    __shared__ float inv_length_cache[BLOCK_SIZE];

    const size_t cache_size = min(npixel - ipix_offset, (size_t)BLOCK_SIZE);
    if (threadIdx.x < cache_size) {
        const size_t ipix = ipix_offset + threadIdx.x;
        measured_cache[threadIdx.x] = measured[ipix];
        fitted_cache[threadIdx.x] = fitted[ipix];
        const float length = ray_length[ipix];
        inv_length_cache[threadIdx.x] = (length > ray_length_thres) ? 1.f / length : 0;
    }

    __syncthreads();

    const float ray_dens = (jvox < nvoxel) ? ray_density[jvox] : 0;
    float obs = 0;
    float fit = 0;
    if (ray_dens > ray_dens_thres) {
        if (cache_size % 4 == 0) {
            float4 prop_ray_length;
            for (unsigned int ipix_cache = 0; ipix_cache < cache_size; ipix_cache += 4) {
                const unsigned int ipix = ipix_offset + ipix_cache;
                prop_ray_length.x = rtm[ ipix      * nvoxel + jvox] * inv_length_cache[ipix_cache    ];
                prop_ray_length.y = rtm[(ipix + 1) * nvoxel + jvox] * inv_length_cache[ipix_cache + 1];
                prop_ray_length.z = rtm[(ipix + 2) * nvoxel + jvox] * inv_length_cache[ipix_cache + 2];
                prop_ray_length.w = rtm[(ipix + 3) * nvoxel + jvox] * inv_length_cache[ipix_cache + 3];
                obs += prop_ray_length.x * measured_cache[ipix_cache    ] + \
                       prop_ray_length.y * measured_cache[ipix_cache + 1] + \
                       prop_ray_length.z * measured_cache[ipix_cache + 2] + \
                       prop_ray_length.w * measured_cache[ipix_cache + 3];
                fit += prop_ray_length.x * fitted_cache[ipix_cache    ] + \
                       prop_ray_length.y * fitted_cache[ipix_cache + 1] + \
                       prop_ray_length.z * fitted_cache[ipix_cache + 2] + \
                       prop_ray_length.w * fitted_cache[ipix_cache + 3];
            }
        }
        else {
            float prop_ray_length;
            for (unsigned int ipix_cache = 0; ipix_cache < cache_size; ++ipix_cache) {
                prop_ray_length = rtm[(ipix_offset + ipix_cache) * nvoxel + jvox] * inv_length_cache[ipix_cache];
                obs += prop_ray_length * measured_cache[ipix_cache];
                fit += prop_ray_length * fitted_cache[ipix_cache];
            }
        }
        atomicAdd(obs_fit + jvox, obs);
        atomicAdd(obs_fit + nvoxel + jvox, fit);
    }
}


__global__ void GradPenaltyKernel(float* const __restrict__ grad_penalty,  const float* const __restrict__ solution,
                                  const size_t* const __restrict__ laplace_idx, const float* const __restrict__ laplace_val,
                                  float beta_laplace, size_t laplacian_size, size_t nvoxel) {

    size_t i_offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t i = i_offset; i < laplacian_size; i += blockDim.x * gridDim.x) {
        const size_t index = laplace_idx[i];
        atomicAdd(grad_penalty + index / nvoxel, beta_laplace * laplace_val[i] * solution[index % nvoxel]);
    }
}


__global__ void LogGradPenaltyKernel(float* const __restrict__ grad_penalty,  const float* const __restrict__ solution,
                                     const size_t* const __restrict__ laplace_idx, const float* const __restrict__ laplace_val,
                                     float beta_laplace, size_t laplacian_size, size_t nvoxel) {

    size_t i_offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(size_t i = i_offset; i < laplacian_size; i += blockDim.x * gridDim.x) {
        const size_t index = laplace_idx[i];
        atomicAdd(grad_penalty + index / nvoxel, beta_laplace * laplace_val[i] * __logf(solution[index % nvoxel]));
    }
}


__global__ void UpdateSolutionKernel(float* const __restrict__ solution, float* const __restrict__ diff, size_t nvoxel){
    size_t jvox = blockIdx.x * blockDim.x + threadIdx.x;
    if (jvox < nvoxel) {
        float sol = solution[jvox] + diff[jvox];
        solution[jvox] = (sol > 0) ? sol : 0;
        diff[jvox] = 0;
    }
}


__global__ void UpdateLogSolutionKernel(float* const __restrict__ solution, float* const __restrict__ obs_fit,
                                        const float* const __restrict__ grad_penalty, float relaxation, size_t nvoxel){
    size_t jvox = blockIdx.x * blockDim.x + threadIdx.x;
    if (jvox < nvoxel) {
        const float ratio = __powf((obs_fit[jvox] + EPSILON_LOG_CUDA) / (obs_fit[nvoxel + jvox] + EPSILON_LOG_CUDA), relaxation);
        solution[jvox] *= ratio * __expf(-grad_penalty[jvox]);
        obs_fit[jvox] = 0;
        obs_fit[nvoxel + jvox] = 0;
    }
}


extern "C" void CallInitialGuessKernel(float* const solution, const float* const rtm, const float* const measured, 
                                       const float* const ray_density, const float ray_dens_thres, const size_t npixel, const size_t nvoxel) {

    dim3 dim_block(BLOCK_SIZE, 1);
    dim3 dim_grid(nvoxel / BLOCK_SIZE + (bool)(nvoxel % BLOCK_SIZE), npixel / BLOCK_SIZE + (bool)(npixel % BLOCK_SIZE));
    InitialGuessKernel<<<dim_grid, dim_block>>>(solution, rtm, measured, ray_density, ray_dens_thres, npixel, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "InitialGuessKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallGradPenaltyKernel(float* const grad_penalty, const float* const solution, const size_t* const laplace_idx,
                                      const float* const laplace_val, float beta_laplace, size_t laplacian_size, size_t nvoxel) {

    size_t dim_grid = laplacian_size / BLOCK_SIZE + (bool)(laplacian_size % BLOCK_SIZE);
    GradPenaltyKernel<<<dim_grid, BLOCK_SIZE>>>(grad_penalty, solution, laplace_idx, laplace_val,
                                                beta_laplace, laplacian_size, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "GradPenaltyKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallLogGradPenaltyKernel(float* const grad_penalty, const float* const solution, const size_t* const laplace_idx,
                                         const float* const laplace_val, float beta_laplace, size_t laplacian_size, size_t nvoxel) {

    size_t dim_grid = laplacian_size / BLOCK_SIZE + (bool)(laplacian_size % BLOCK_SIZE);
    LogGradPenaltyKernel<<<dim_grid, BLOCK_SIZE>>>(grad_penalty, solution, laplace_idx, laplace_val,
                                                   beta_laplace, laplacian_size, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "LogGradPenaltyKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallPropagateKernel(float* const diff, const float* const rtm, const float* const measured, const float* const fitted,
                                    const float* const ray_density, const float* const ray_length, const float* const grad_penalty,
                                    float relaxation, float ray_dens_thres, float ray_length_thres, size_t npixel, size_t nvoxel) {

    dim3 dim_block(BLOCK_SIZE, 1);
    dim3 dim_grid(nvoxel / BLOCK_SIZE + (bool)(nvoxel % BLOCK_SIZE), npixel / BLOCK_SIZE + (bool)(npixel % BLOCK_SIZE));
    PropagateKernel<<<dim_grid, dim_block>>>(diff, rtm, measured, fitted, ray_density, ray_length, grad_penalty,
                                             relaxation, ray_dens_thres, ray_length_thres, npixel, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "PropagateKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallLogPropagateKernel(float* const ofs_fit, const float* const rtm, const float* const measured, const float* const fitted,
                                       const float* const ray_density, const float* const ray_length,
                                       float ray_dens_thres, float ray_length_thres, size_t npixel, size_t nvoxel) {

    dim3 dim_block(BLOCK_SIZE, 1);
    dim3 dim_grid(nvoxel / BLOCK_SIZE + (bool)(nvoxel % BLOCK_SIZE), npixel / BLOCK_SIZE + (bool)(npixel % BLOCK_SIZE));
    LogPropagateKernel<<<dim_grid, dim_block>>>(ofs_fit, rtm, measured, fitted, ray_density, ray_length,
                                                ray_dens_thres, ray_length_thres, npixel, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "LogPropagateKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallUpdateSolutionKernel(float* const solution, float* const diff, size_t nvoxel) {

    size_t dim_grid = nvoxel / BLOCK_SIZE + (bool)(nvoxel % BLOCK_SIZE);
    UpdateSolutionKernel<<<dim_grid, BLOCK_SIZE>>>(solution, diff, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "UpdateSolutionKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}

extern "C" void CallUpdateLogSolutionKernel(float* const solution, float* const ofs_fit, const float* const grad_penalty,
                                            float relaxation, size_t nvoxel) {

    size_t dim_grid = nvoxel / BLOCK_SIZE + (bool)(nvoxel % BLOCK_SIZE);
    UpdateLogSolutionKernel<<<dim_grid, BLOCK_SIZE>>>(solution, ofs_fit, grad_penalty, relaxation, nvoxel);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "UpdateLogSolutionKernel<<<,>>>(...) failure:\n";
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(1);
    }
}
